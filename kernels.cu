#include "hip/hip_runtime.h"
#include "kernels.h"
#include "gaussian.h"
#include <iostream>
#include <cmath>


__global__
void blur(unsigned char* input_image, unsigned char* output_image, int width, int height) {

    const unsigned int offset = blockIdx.x*blockDim.x + threadIdx.x;// numer pixela
    int x = offset % width;
    int y = offset/width;
    int fsize = 5; // Filter size
    if(offset < width*height) {

        float output_red = 0;
        float output_green = 0;
        float output_blue = 0;
        int hits = 0;
        for(int ox = -fsize; ox < fsize+1; ++ox) {
            for(int oy = -fsize; oy < fsize+1; ++oy) {
                if((x+ox) > -1 && (x+ox) < width && (y+oy) > -1 && (y+oy) < height) {
                    const int currentoffset = (offset+ox+oy*width)*3;
                    output_red += input_image[currentoffset]; 
                    output_green += input_image[currentoffset+1];
                    output_blue += input_image[currentoffset+2];
                    hits++;
                }
            }
        }
        output_image[offset*3] = output_red/hits;
        output_image[offset*3+1] = output_green/hits;
        output_image[offset*3+2] = output_blue/hits;
        }
}


void filter (unsigned char* input_image, unsigned char* output_image, int width, int height) {

    unsigned char* dev_input;
    unsigned char* dev_output;
    getError(hipMalloc( (void**) &dev_input, width*height*3*sizeof(unsigned char)));
    getError(hipMemcpy( dev_input, input_image, width*height*3*sizeof(unsigned char), hipMemcpyHostToDevice ));
 
    getError(hipMalloc( (void**) &dev_output, width*height*3*sizeof(unsigned char)));

    //dim3 blockDims(512,1,1);
    //dim3 gridDims((unsigned int) ceil((double)(width*height*3/blockDims.x)), 1, 1 );

    //--- run kernel ---
    dim3 numThreads = dim3(BLOCKSIZE_X, BLOCKSIZE_Y, 1);
    dim3 numBlocks = dim3(width / numThreads.x, height / numThreads.y);

    // First run the warmup kernel (which we'll use to get the GPU in the correct max power state
    blur<<<numBlocks, numThreads>>>(dev_input, dev_output, width, height); 
    hipDeviceSynchronize();

    //--- Allocate CUDA events that we'll use for timing ---
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));

    printf("Launching CUDA Kernel\n");

    //--- Record the start event ---
    checkCudaErrors(hipEventRecord(start, NULL));

    blur<<<numBlocks, numThreads>>>(dev_input, dev_output, width, height);
    //--- Record the stop event ---
    checkCudaErrors(hipEventRecord(stop, NULL));

    //--- Wait for the stop event to complete ---
    checkCudaErrors(hipEventSynchronize(stop));

    //--- Check to make sure the kernel didn't fail ---
    getLastCudaError("Kernel execution failed");

    float msecTotal = 0.0f;
    checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));

    getError(hipMemcpy(output_image, dev_output, width*height*3*sizeof(unsigned char), hipMemcpyDeviceToHost ));

    getError(hipFree(dev_input));
    getError(hipFree(dev_output));

}

